#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

// #include "modules/drug_sim.hpp"
#include <math.h>
#include <sys/stat.h>

#include <algorithm>
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <ctime>
#include <filesystem>
#include <iostream>
#include <regex>
#include <string>
#include <unordered_map>
#include <vector>

#include "modules/cipa_t.cuh"
#include "modules/glob_funct.hpp"
#include "modules/glob_type.hpp"
#include "modules/gpu.cuh"
namespace fs = std::filesystem;

#define ENOUGH ((CHAR_BIT * sizeof(int) - 1) / 3 + 3)
char buffer[255];

// unsigned int datapoint_size = 7000;
const unsigned int sample_limit = 10000;

clock_t START_TIMER;

clock_t tic();
void toc(clock_t start = START_TIMER);

clock_t tic() { return START_TIMER = clock(); }

void toc(clock_t start) {
    std::cout << "Elapsed time: " << (clock() - start) / (double)CLOCKS_PER_SEC << "s" << std::endl;
}

int gpu_check(unsigned int datasize) {
    int num_gpus;
    float percent;
    int id;
    size_t free, total;
    hipGetDeviceCount(&num_gpus);
    for (int gpu_id = 0; gpu_id < num_gpus; gpu_id++) {
        hipSetDevice(gpu_id);
        hipGetDevice(&id);
        hipMemGetInfo(&free, &total);
        percent = (free / (float)total);
        printf("GPU No %d\nFree Memory: %ld, Total Memory: %ld (%f percent free)\n", id, free, total, percent * 100.0);
    }
    percent = 1.0 - (datasize / (float)total);
    //// this code strangely gave out too small value, so i disable the safety switch for now

    // printf("The program uses GPU No %d and %f percent of its memory\n", id,percent*100.0);
    // printf("\n");
    // if (datasize<=free) {
    //   return 0;
    // }
    // else {
    //   return 1;
    // }

    return 0;
}

// get the IC50 data from file
drug_t get_IC50_data_from_file(const char *file_name);
// return error and message based on the IC50 data

void addDrugData(char ***arrayOfStrings, int &size, const char newString[]) {
    char **newArray = new char *[size + 1];

    // Copy existing strings to the new array
    for (int i = 0; i < size; ++i) {
        newArray[i] = new char[strlen((*arrayOfStrings)[i]) + 1];
        strcpy(newArray[i], (*arrayOfStrings)[i]);
        delete[] (*arrayOfStrings)[i];  // Deallocate memory for old strings
    }

    // Allocate memory for the new string and copy it
    newArray[size] = new char[strlen(newString) + 1];
    strcpy(newArray[size], newString);

    // Deallocate memory for the old array
    delete[] *arrayOfStrings;

    // Update the pointer to point to the new array
    *arrayOfStrings = newArray;

    // Increment the size
    ++size;
}

int check_IC50_content(const drug_t *ic50, const param_t *p_param);

int get_IC50_data_from_file(const char *file_name, double *ic50) {
    /*
    a host function to take all samples from the file, assuming each sample has 14 features.

    it takes the file name, and an ic50 (already declared in 1D, everything become 1D)
    as a note, the data will be stored in 1D array, means this functions applies flatten.

    it returns 'how many samples were detected?' in integer.
    */
    FILE *fp_drugs;
    //   drug_t ic50;
    char *token;
    char buffer_ic50[255];
    unsigned int idx;

    if ((fp_drugs = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n", file_name);
        return 0;
    }
    idx = 0;
    int sample_size = 0;
    fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs);                   // skip header
    while (fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs) != NULL) {  // begin line reading
        token = strtok(buffer_ic50, ",");
        while (token != NULL) {  // begin data tokenizing
            ic50[idx++] = strtod(token, NULL);
            token = strtok(NULL, ",");
        }  // end data tokenizing
        sample_size++;
    }  // end line reading

    fclose(fp_drugs);
    return sample_size;
}

int get_IC50_data_from_file(const char *file_name, double *ic50, double *conc, char **drug_name) {
    /*
    a host function to take all samples from the file, assuming each sample has 14 features.

    it takes the file name, and an ic50 (already declared in 1D, everything become 1D)
    as a note, the data will be stored in 1D array, means this functions applies flatten.

    it returns 'how many samples were detected?' in integer.
    */
    FILE *fp_drugs;
    //   drug_t ic50;
    char *token;
    char tmp_drug_name[32];
    char buffer_ic50[255];
    unsigned int idx_ic50, idx_conc;
    int drugsize = 0;

    if ((fp_drugs = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n", file_name);
        return 0;
    }
    idx_ic50 = 0;
    idx_conc = 0;
    int sample_size = 0;
    fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs);                   // skip header
    while (fgets(buffer_ic50, sizeof(buffer_ic50), fp_drugs) != NULL) {  // begin line reading
        /*
        TODO: Extracting token from file
        1. take token for each file
        2. check the first token to drug_name, if already exist in array, then skip it
        3. check the second token to conc
        */

        token = strtok(buffer_ic50, ",");
        // printf("%s\n", token);  // testingAuto
        strcpy(tmp_drug_name, token);
        token = strtok(NULL, ",");
        // printf("%s\n", token);  // testingAuto
        strcat(tmp_drug_name, "_");
        strcat(tmp_drug_name, token);

        // printf("%s\n", tmp_drug_name);  // testingAuto
        addDrugData(&drug_name, drugsize, tmp_drug_name);
        conc[idx_conc++] = strtod(token, NULL);
        token = strtok(NULL, ",");
        // Check if there is wrong in here
        while (token != NULL) {  // begin data tokenizing
            ic50[idx_ic50++] = strtod(token, NULL);
            // printf("%s\n", token);  // testingAuto
            token = strtok(NULL, ",");
        }  // end data tokenizing
        sample_size++;
    }  // end line reading

    fclose(fp_drugs);
    return sample_size;
}

int get_cvar_data_from_file(const char *file_name, unsigned int limit, double *cvar) {
    // buffer for writing in snprintf() function
    char buffer_cvar[255];
    FILE *fp_cvar;
    // cvar_t cvar;
    char *token;
    // std::array<double,18> temp_array;
    unsigned int idx;

    if ((fp_cvar = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n", file_name);
    }
    idx = 0;
    int sample_size = 0;
    fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar);                                              // skip header
    while ((fgets(buffer_cvar, sizeof(buffer_cvar), fp_cvar) != NULL) && (sample_size < limit)) {  // begin line reading
        token = strtok(buffer_cvar, ",");
        while (token != NULL) {  // begin data tokenizing
            cvar[idx++] = strtod(token, NULL);
            token = strtok(NULL, ",");
        }  // end data tokenizing
        // printf("\n");
        sample_size++;
        // cvar.push_back(temp_array);
    }  // end line reading

    fclose(fp_cvar);
    return sample_size;
}

int get_init_data_from_file(const char *file_name, double *init_states) {
    // buffer for writing in snprintf() function
    char buffer_cache[1023];
    FILE *fp_cache;
    // cvar_t cvar;
    char *token;
    // std::array<double,18> temp_array;
    unsigned long idx;

    if ((fp_cache = fopen(file_name, "r")) == NULL) {
        printf("Cannot open file %s\n", file_name);
    }
    idx = 0;
    unsigned int sample_size = 0;
    fgets(buffer_cache, sizeof(buffer_cache), fp_cache);                     // skip header
    while ((fgets(buffer_cache, sizeof(buffer_cache), fp_cache) != NULL)) {  // begin line reading
        token = strtok(buffer_cache, ",");
        while (token != NULL) {  // begin data tokenizing
            init_states[idx++] = strtod(token, NULL);
            // if(idx < 82){
            //     printf("%d: %lf\n",idx-1,init_states[idx-1]);
            // }
            token = strtok(NULL, ",");
        }  // end data tokenizing
        // printf("\n");
        sample_size++;
        // cvar.push_back(temp_array);
    }  // end line reading

    fclose(fp_cache);
    return sample_size;
}

int exists(const char *fname) {
    FILE *file;
    if ((file = fopen(fname, "r"))) {
        fclose(file);
        return 1;
    }
    // fclose(file);
    return 0;
}

int check_IC50_content(const drug_t *ic50, const param_t *p_param) {
    if (ic50->size() == 0) {
        printf("Something problem with the IC50 file!\n");
        return 1;
    } else if (ic50->size() > 2000) {
        printf("Too much input! Maximum sample data is 2000!\n");
        return 2;
    } else if (p_param->pace_max < 750 && p_param->pace_max > 1000) {
        printf("Make sure the maximum pace is around 750 to 1000!\n");
        return 3;
    }
    // else if(mympi::size > ic50->size()){
    // 	printf("%s\n%s\n",
    //               "Overflow of MPI Process!",
    //               "Make sure MPI Size is less than or equal the number of sample");
    // 	return 4;
    // }
    else {
        return 0;
    }
}

int get_herg_data_from_file(const char* file_name, double *herg)
{
  FILE *fp_herg;
  char *token;
  char buffer_herg[255];
  unsigned int idx;

  if( (fp_herg = fopen(file_name, "r")) == NULL){
    printf("Cannot open file %s\n", file_name);
    return 0;
  }
  printf("Herg file %s\n", file_name);
  idx = 0;
  int sample_size = 0;
  fgets(buffer_herg, sizeof(buffer_herg), fp_herg); // skip header
  while( fgets(buffer_herg, sizeof(buffer_herg), fp_herg) != NULL )
    { // begin line reading
      token = strtok( buffer_herg, "," );
      while( token != NULL )
      { // begin data tokenizing
        herg[idx++] = strtod(token, NULL);
        token = strtok(NULL, ",");
      } // end data tokenizing
      sample_size++;
    } // end line reading

  fclose(fp_herg);
  return sample_size;
}

int main(int argc, char **argv) {
    // enable real-time output in stdout
    setvbuf(stdout, NULL, _IONBF, 0);

    // NEW CODE STARTS HERE //
    double inal_auc_control = -90.547322;    // AUC of INaL under control model
    double ical_auc_control = -105.935067;   // AUC of ICaL under control model
    // mycuda *thread_id;
    // hipMalloc(&thread_id, sizeof(mycuda));

    // input variables for cell simulation
    param_t *p_param, *d_p_param;
    p_param = new param_t();
    p_param->init();
    edison_assign_params(argc, argv, p_param);
    p_param->show_val();

    std::regex pattern("/([a-zA-Z0-9_\.]+)\.csv");
    std::smatch match;
    std::string fname = p_param->hill_file;
    std::regex_search(fname, match, pattern);

    printf("%s\n", match[1].str().c_str());

    double *ic50;  // temporary
    double *cvar;
    double *conc;
    char **drug_name = nullptr;

    ic50 = (double *)malloc(14 * sample_limit * sizeof(double));
    // if (p_param->is_cvar == true) cvar = (double *)malloc(18 * sample_limit * sizeof(double));
    cvar = (double *)malloc(18 * sample_limit * sizeof(double));
    conc = (double *)malloc(sample_limit * sizeof(double));
    double* herg = (double *)malloc(6 * sample_limit * sizeof(double));

    int num_of_constants = 208;
    int num_of_states = 49;
    int num_of_algebraic = 200;
    int num_of_rates = 49;

    // const double CONC = p_param->conc;

    //////// if we are in write time series mode (post processing) //////////

    printf("Using cached initial state from previous result!!!! \n\n");

    unsigned int datapoint_size;
    bool is_euler = true;

    if(is_euler){
        // datapoint_size = p_param->bcl / p_param->dt;
        datapoint_size = 1001;
    }
    else {
        datapoint_size = p_param->sampling_limit;
    }

    double *cache;
    cache = (double *)malloc(num_of_states * sample_limit * sizeof(double));

    double *d_ic50;
    double *d_conc;
    double *d_cvar;
    double *d_ALGEBRAIC;
    double *d_CONSTANTS;
    double *d_RATES;
    double *d_STATES;
    double *d_STATES_cache;

    // actually not used but for now, this is only for satisfiying the GPU regulator parameters
    double *d_STATES_RESULT;
    double *d_all_states;

    double *time;
    double *dt;
    double *states;
    double *ical;
    double *inal;
    double *cai_result;
    double *ina;
    double *ito;
    double *ikr;
    double *iks;
    double *ik1;
    cipa_t *temp_result, *cipa_result;

    static const int CALCIUM_SCALING = 1000000;
    static const int CURRENT_SCALING = 1000;

    // snprintf(buffer, sizeof(buffer),
    //   "./drugs/bepridil/IC50_samples.csv"
    //   // "./drugs/bepridil/IC50_optimal.csv"
    //   // "./IC50_samples.csv"
    //   );

    int sample_size = get_IC50_data_from_file(p_param->hill_file, ic50, conc, drug_name);
    if (sample_size == 0) printf("Something problem with the IC50 file!\n");
    // else if(sample_size > 2000)
    //     printf("Too much input! Maximum sample data is 2000!\n");
    printf("Sample size: %d\n", sample_size);
    printf("Set GPU Number: %d\n", p_param->gpu_index);

    hipSetDevice(p_param->gpu_index);

    if (p_param->is_cvar == true) {
        int cvar_sample = get_cvar_data_from_file(p_param->cvar_file, sample_size, cvar);
        printf("Reading: %d Conductance Variability samples\n", cvar_sample);
    }

    int herg_size = get_herg_data_from_file(p_param->herg_file, herg);
        if(herg_size == 0)
            printf("Something problem with the herg file!\n");
        
        printf("herg size: %d herg check:\n", herg_size);
            for(int temp = 0; temp<6; temp++){
            printf("%lf, ",herg[temp]);
            } 
            printf("\n");

    printf("preparing GPU memory space \n");

    int cache_num = get_init_data_from_file(p_param->cache_file, cache);
    printf("Found cache for %d samples\n", cache_num);

    hipMalloc(&d_ALGEBRAIC, num_of_algebraic * sample_size * sizeof(double));
    hipMalloc(&d_CONSTANTS, num_of_constants * sample_size * sizeof(double));
    hipMalloc(&d_RATES, num_of_rates * sample_size * sizeof(double));
    hipMalloc(&d_STATES, num_of_states * sample_size * sizeof(double));
    hipMalloc(&d_STATES_cache, (num_of_states + 2) * sample_size * sizeof(double));

    hipMalloc(&d_p_param, sizeof(param_t));

    // prep for 1 cycle plus a bit (7000 * sample_size)
    hipMalloc(&temp_result, sample_size * sizeof(cipa_t));
    hipMalloc(&cipa_result, sample_size * sizeof(cipa_t));

    hipMalloc(&time, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&dt, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&states, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ical, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&inal, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&cai_result, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ina, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ito, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ikr, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&iks, sample_size * datapoint_size * sizeof(double));
    hipMalloc(&ik1, sample_size * datapoint_size * sizeof(double));
    // hipMalloc(&d_STATES_RESULT, (num_of_states+1) * sample_size * sizeof(double));
    // hipMalloc(&d_all_states, num_of_states * sample_size * p_param->find_steepest_start * sizeof(double));

    printf("Copying sample files to GPU memory space \n");
    hipMalloc(&d_ic50, sample_size * 14 * sizeof(double));
    hipMalloc(&d_cvar, sample_size * 18 * sizeof(double));
    hipMalloc(&d_conc, sample_size * sizeof(double));
    
    hipMemcpy(d_STATES_cache, cache, (num_of_states + 2) * sample_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_ic50, ic50, sample_size * 14 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_cvar, cvar, sample_size * 18 * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_conc, conc, sample_size * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(d_p_param, p_param, sizeof(param_t), hipMemcpyHostToDevice);

      // for BDF
        //double *d_all_states;
        double *d_herg;

        double *y; double *y_new; double *F; double *delta; double *Jc; 
        double *y_perturbed; double *g0; double *g_perturbed; 
        hipMalloc(&y, num_of_states * sample_size * sizeof(double));
        hipMalloc(&y_new, num_of_states * sample_size * sizeof(double));
        hipMalloc(&F, num_of_states * sample_size * sizeof(double));
        hipMalloc(&delta, num_of_states * sample_size * sizeof(double));
        hipMalloc(&Jc, num_of_states * num_of_states * sample_size * sizeof(double));

        hipMalloc(&y_perturbed, num_of_states * sample_size * sizeof(double));
        hipMalloc(&g0, num_of_states * sample_size * sizeof(double));
        hipMalloc(&g_perturbed, num_of_states * sample_size * sizeof(double));

        hipMalloc(&d_all_states, num_of_states * sample_size * p_param->find_steepest_start * sizeof(double)); // for each sample
        hipMalloc(&d_herg, 6 * sample_size * sizeof(double));
        hipMemcpy(d_herg, herg, sample_size * 6 * sizeof(double), hipMemcpyHostToDevice);

    // // Get the maximum number of active blocks per multiprocessor
    // hipOccupancyMaxActiveBlocksPerMultiprocessor(&numBlocks, do_drug_sim_analytical, threadsPerBlock);

    // // Calculate the total number of blocks
    // int numTotalBlocks = numBlocks * cudaDeviceGetMultiprocessorCount();

    tic();
    printf("Timer started, doing simulation.... \n\n\nGPU Usage at this moment: \n");
    int thread = 20;
    int block = (sample_size + thread - 1) / thread;
    // int block = (sample_size + thread - 1) / thread;
    if (gpu_check(15 * sample_size * sizeof(double) + sizeof(param_t)) == 1) {
        printf("GPU memory insufficient!\n");
        return 0;
    }
    printf("Sample size: %d\n", sample_size);
    hipSetDevice(p_param->gpu_index);
    printf("\n   Configuration: \n\n\tblock\t||\tthread\n---------------------------------------\n  \t%d\t||\t%d\n\n\n",
           block, thread);
    // initscr();
    // printf("[____________________________________________________________________________________________________]
    // 0.00 %% \n");
    // (double *d_ic50, double *d_cvar, double *d_conc, double *d_CONSTANTS,
    //                                   double *d_STATES, double *d_STATES_cache, double *d_RATES, double *d_ALGEBRAIC,
    //                                   double *d_STATES_RESULT, double *d_all_states, double *d_herg,
    //                                   double *time, double *states,
    //                                   double *out_dt, double *cai_result, double *ina, double *inal, double *ical,
    //                                   double *ito, double *ikr, double *iks, double *ik1, unsigned int sample_size,
    //                                   cipa_t *temp_result, cipa_t *cipa_result, param_t *p_param,
    //                                   double *y, double *y_new, double *F, double *delta, double *Jc, double *y_perturbed, double *g0, double *g_perturbed) 

    kernel_DrugSimulation<<<block, thread>>>(d_ic50, d_cvar, d_conc, d_CONSTANTS, d_STATES, d_STATES_cache, d_RATES,
                                             d_ALGEBRAIC, d_STATES_RESULT, d_all_states, d_herg, time, states, dt, cai_result,
                                             ina, inal, ical, ito, ikr, iks, ik1, sample_size, temp_result, cipa_result,
                                             d_p_param, y, y_new, F, delta, Jc, y_perturbed, g0, g_perturbed);
    // block per grid, threads per block
    // endwin();

    hipDeviceSynchronize();

    printf("allocating memory for computation result in the CPU, malloc style \n");
    double *h_states, *h_time, *h_dt, *h_ical, *h_inal, *h_cai_result, *h_ina, *h_ito, *h_ikr, *h_iks, *h_ik1;
    cipa_t *h_cipa_result;

    h_states = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for STATES, \n");
    h_time = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for time, \n");
    h_dt = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for dt, \n");
    h_cai_result = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for Cai, \n");
    h_ina = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for iNa, \n");
    h_ito = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ito, \n");
    h_ikr = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ikr, \n");
    h_iks = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for iks, \n");
    h_ik1 = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ik1, \n");
    h_ical = (double *)malloc(datapoint_size * sample_size * sizeof(double));
    printf("...allocated for ICaL, \n");
    h_inal = (double *)malloc(datapoint_size * sample_size * sizeof(double));

    h_cipa_result = (cipa_t *)malloc(sample_size * sizeof(cipa_t));
    printf("...allocating for INaL and postprocessing, all set!\n");

    ////// copy the data back to CPU, and write them into file ////////
    printf("copying the data back to the CPU \n");

    hipMemcpy(h_states, states, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_time, time, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_dt, dt, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ical, ical, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_inal, inal, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_cai_result, cai_result, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ina, ina, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ito, ito, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ikr, ikr, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_iks, iks, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);
    hipMemcpy(h_ik1, ik1, sample_size * datapoint_size * sizeof(double), hipMemcpyDeviceToHost);

    hipMemcpy(h_cipa_result, cipa_result, sample_size * sizeof(cipa_t), hipMemcpyDeviceToHost);

    hipFree(d_ALGEBRAIC);
    hipFree(d_CONSTANTS);
    hipFree(d_RATES);
    hipFree(d_STATES);
    hipFree(d_p_param);
    hipFree(temp_result);
    hipFree(cipa_result);
    hipFree(d_STATES_RESULT);
    hipFree(d_ic50);
    hipFree(d_cvar);
    hipFree(d_conc);
    hipFree(time);
    hipFree(dt);
    hipFree(states);
    hipFree(ical);
    hipFree(inal);
    hipFree(cai_result);
    hipFree(ina);
    hipFree(ito);
    hipFree(ikr);
    hipFree(iks);
    hipFree(ik1);

    FILE *writer;
    int check;
    bool folder_created = false;

    printf("writing to file... \n");
    // sample loop
    for (int sample_id = 0; sample_id < sample_size; sample_id++) {
        // printf("writing sample %d... \n",sample_id);
        char sample_str[ENOUGH];
        char conc_str[ENOUGH];
        char filename[500] = "./result/post_";
        sprintf(sample_str, "%d", sample_id);
        // sprintf(conc_str, "%.2f", conc[sample_id]);
        strcat(filename, match[1].str().c_str());
        strcat(filename, "/");
        if (folder_created == false) {
            check = mkdir(filename, 0777);
            // check if directory is created or not
            if (!check) {
                printf("Directory created\n");
            } else {
                printf("Unable to create directory, or the folder is already created, relax mate...\n");
            }
            folder_created = true;
        }

        strcat(filename, sample_str);
        strcat(filename, "_pace.csv");

        writer = fopen(filename, "w");
        fprintf(writer, "Time,Vm,dVm/dt,Cai,INa,INaL,ICaL,IKs,IKr,IK1,Ito\n");
        for (int datapoint = 1; datapoint < datapoint_size; datapoint++) {
            if (h_time[sample_id + (datapoint * sample_size)] == 0.0) {
                break;
            }
            fprintf(writer,
                    "%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n",  // change this into string, or limit the decimal
                                                                      // accuracy, so we can decrease filesize
                    h_time[sample_id + (datapoint * sample_size)], h_states[sample_id + (datapoint * sample_size)],
                    h_dt[sample_id + (datapoint * sample_size)], h_cai_result[sample_id + (datapoint * sample_size)],

                    h_ina[sample_id + (datapoint * sample_size)], h_inal[sample_id + (datapoint * sample_size)],

                    h_ical[sample_id + (datapoint * sample_size)], h_iks[sample_id + (datapoint * sample_size)],

                    h_ikr[sample_id + (datapoint * sample_size)], h_ik1[sample_id + (datapoint * sample_size)],

                    h_ito[sample_id + (datapoint * sample_size)]);
        }
        fclose(writer);
    }

    printf("writing each biomarkers value... \n");
    // sample loop
    // char conc_str[ENOUGH];
    char filename[500] = "./result/post_";
    // sprintf(sample_str, "%d", sample_id);
    // sprintf(conc_str, "%.2f", conc[sample_id]);
    strcat(filename, match[1].str().c_str());
    strcat(filename, "/");
    // printf("creating %s... \n", filename);
    if (folder_created == false) {
        check = mkdir(filename, 0777);
        // check if directory is created or not
        if (!check) {
            printf("Directory created\n");
        } else {
            printf("Unable to create directory, or the folder is already created, relax mate...\n");
        }
        folder_created = true;
    }

    // strcat(filename,sample_str);
    strcat(filename, "_biomarkers.csv");

    writer = fopen(filename, "a");

    fprintf(writer,"sample,qnet,inal_auc,ical_auc,apd90,apd50,apd_tri,cad90,cad50,cad_tri,dvmdt_repol,dvmdt_peak,vm_peak,vm_valley,vm_dia,ca_peak,ca_valley,ca_dia\n");
    for (int sample_id = 0; sample_id < sample_size; sample_id++) {
        // printf("writing sample %d... \n",sample_id);

        fprintf(writer,
                "%d,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf,%lf\n",  // change this into string, or
                                                                                         // limit the decimal accuracy,
                                                                                         // so we can decrease filesize
                sample_id, h_cipa_result[sample_id].qnet, h_cipa_result[sample_id].inal_auc,
                h_cipa_result[sample_id].ical_auc,

                h_cipa_result[sample_id].apd90, h_cipa_result[sample_id].apd50,
                h_cipa_result[sample_id].apd90 - h_cipa_result[sample_id].apd50,

                h_cipa_result[sample_id].cad90, h_cipa_result[sample_id].cad50,
                h_cipa_result[sample_id].cad90 - h_cipa_result[sample_id].cad50,

                h_cipa_result[sample_id].dvmdt_repol, h_cipa_result[sample_id].dvmdt_max, h_cipa_result[sample_id].vm_peak,
                h_cipa_result[sample_id].vm_valley, h_cipa_result[sample_id].vm_dia,

                h_cipa_result[sample_id].ca_peak, h_cipa_result[sample_id].ca_valley, 
                h_cipa_result[sample_id].ca_dia
                

        );
    }
    fclose(writer);

    toc();

    return 0;
}
